
#include <hip/hip_runtime.h>
/* shapes

 Corners  Edges  Across    TB      LR
  x o x   o x o   x o x   o x o   o o o
  o o o   x o x   o x o   o o o   x o x
  x o x   o x o   x o x   o x o   o o o  */

#define CENTER         (     bayer[i]                                                                          )
#define CORNERS        (med4(bayer[(i-1)-w],  bayer[(i+1)-w],  bayer[(i-1)+w],  bayer[(i+1)+w]           )     )
#define EDGES          (med4(bayer[i-w],      bayer[i-1],      bayer[i+1],      bayer[i+w]               )     )
#define ACROSS         (med5(bayer[(i-1)-w],  bayer[(i+1)-w],  bayer[(i-1)+w],  bayer[(i+1)+w],  bayer[i])     )
#define TOP_AND_BOTTOM (    (bayer[i-w]     + bayer[i+w]                                                 ) >> 1)
#define LEFT_AND_RIGHT (    (bayer[i-1]     + bayer[i+1]                                                 ) >> 1)

#define   RED(shape)  rgb[j+0] = shape
#define GREEN(shape)  rgb[j+1] = shape
#define  BLUE(shape)  rgb[j+2] = shape

__device__ unsigned char med4(unsigned char a, unsigned char b, unsigned char c, unsigned char d) {
	unsigned char l1, l2, h1, h2, t;
	if (a < b) {
		l1 = a;
		h1 = b;
	} else {
		l1 = b;
		h1 = a;
	}

	if (c < d) {
		l2 = c;
		h2 = d;
	} else {
		l2 = d;
		h2 = c;
	}

	if (h2 < h1) {
		t = h2;
		h2 = h1;
		h1 = t;
	}

	if (l2 < l1) {
		t = l2;
		l2 = l1;
		l1 = t;
	}

	return (unsigned char)(((unsigned short)l2 + (unsigned short)h1) >> 1);
}

__device__ unsigned char med5(unsigned char a, unsigned char b, unsigned char c, unsigned char d, unsigned char e) {
	unsigned char l1, l2, h1, h2, t;
	if (a < b) {
		l1 = a;
		h1 = b;
	} else {
		l1 = b;
		h1 = a;
	}

	if (c < d) {
		l2 = c;
		h2 = d;
	} else {
		l2 = d;
		h2 = c;
	}

	if (h2 < h1) {
		t = h2;
		h2 = h1;
		h1 = t;
	}

	if (l2 < l1) {
		t = l2;
		l2 = l1;
		l1 = t;
	}

	if (e <= l2) {
		// e  l1 l2 h1 h2
		// l1  e l2 h1 h2
		// l1 l2  e h1 h2
		return e;
	}

	if (e >= h1) {
		// l1 l2 e  h1 h2
		// l1 l2 h1 e  h2
		// l1 l2 h1 h2 e
		return e;
	}

	return e;
}

extern "C" __global__ void debayer(char *buf, int w, int h) {
	unsigned char *bayer = (unsigned char*)buf;
	unsigned char *rgb   = (unsigned char*)&buf[w*h];
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int i = ((y*w)+x); // offset into bayer
	int j = i*3;       // offset into rgb

	// skip edges
	if (y < 1 || y >= (h-1) || x < 1 || x >= (w-1))
		return;

	if ((1 & x & y) == 1) {
		RED(CORNERS); GREEN(EDGES); BLUE(CENTER);
	} else if ((x & 1) == 1 && (y & 1) == 0) {
		RED(LEFT_AND_RIGHT); GREEN(ACROSS); BLUE(TOP_AND_BOTTOM);
	} else if (/*x % 2 == 0 &&*/ (y & 1) == 1) {
		RED(TOP_AND_BOTTOM); GREEN(ACROSS); BLUE(LEFT_AND_RIGHT);
	} else /*if (x % 2 == 0 && y % 2 == 0) */ {
		RED(CENTER); GREEN(EDGES); BLUE(CORNERS);
	}
}
